//2-opt in cuda


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>


__global__ void kernel(int *array)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  array[index] = 7;
}
