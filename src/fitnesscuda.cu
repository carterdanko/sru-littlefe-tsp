#include "hip/hip_runtime.h"
/** Author: Mike Tasota
 *  Date:   18 September 2012
 *  Descr:  Pretending like this was never accidentally lost the first time
 *          and that I didn't have to rewrite this code again. OK? So, this
 *          is the original file I wrote, because I only wrote this once.
 */
#include <stdio.h>
#include <math.h>
#include "include/fitness.h"
#include "include/tsp.h"
#include "util.c"
//~~!
#include "tsp.c"

#define DEBUG 1

/** Stores distances from every point to another. */
//float distTable[TABLE_SIZE];


/**
 * Using The Pythagorean's Theorem, calculate the distance from p1 to p2. (CUDA)
 */
__device__ float dev_get_distance_between(city_t* citylist) {
	float x,y;
	x = citylist[threadIdx.x].x - citylist[blockIdx.x].x;
	y = citylist[threadIdx.x].y - citylist[blockIdx.x].y;
	return hypotf(x,y); // CUDA function.
}

/**
 * CUDA function for finding table distances.
 */
__global__ void compute_distances(float *table, city_t *cities) {
	if (threadIdx.x < blockIdx.x) {
		int index = blockIdx.x * (blockIdx.x - 1) / 2;
		index+= threadIdx.x;
		table[index] = dev_get_distance_between(cities);
	}
}



/**
 * Using The Pythagorean's Theorem, calculate the distance from p1 to p2.
 */
float get_distance_between(int p1, int p2, tour_t* cities) {
	float x,y;
	x = cities->city[p1]->x - cities->city[p2]->x;
	y = cities->city[p1]->y - cities->city[p2]->y;
	return sqrtf(x*x+y*y);
}

/**
 *  Constructs the distTable. Implemented with CUDA.
 */
void construct_distTable(tour_t* cities, int num_cities) {
	int num_bytes = TABLE_SIZE * sizeof(float);
	int i;
	float *dev_table;
	city_t *dev_cities;//,*host_cities;
	city_t host_cities[MAX_CITIES];

	for (i=0;i<num_cities;i++) {
		host_cities[i]=*cities->city[i];
	}
	memset((void*)distTable, 0, sizeof(distTable));

	// allocate memory for cuda device variable
	hipMalloc((void**)&dev_table,num_bytes);
	hipMalloc((void**)&dev_cities,sizeof(host_cities));
	hipMemcpy(dev_cities,host_cities,sizeof(host_cities),hipMemcpyHostToDevice);
	hipMemcpy(dev_table,distTable,sizeof(distTable),hipMemcpyHostToDevice);

	// now, pass the table through to the GPU.
	compute_distances<<<num_cities,num_cities>>>(dev_table,dev_cities);

	// Now, read memory back to host
	hipMemcpy(distTable,dev_table,num_bytes,hipMemcpyDeviceToHost);

	// deallocate memory
	hipFree(dev_table);
	hipFree(dev_cities);
}

/**
 * Returns the distance traveled from a point p1 to another point p2.
 *   This is retrieved from the distTable hashtable.
 */
float lookup_distance(int p1, int p2) {
        if (p1<p2) {
                return distTable[(p2*(p2-1)/2)+p1];
        } else if (p1>p2) {
                return distTable[(p1*(p1-1)/2)+p2];
        } else {
                printf("WARNING -- THIS SHOULD NEVER HAPPEN (p1==p2); returning 0...\n");
                return 0.0;
        }
}


/**
 * Given a tour and the number of cities, determine its fitness by
 * computing the distance required to traverse the route.
 */
void set_tour_fitness(tour_t* tour, int num_cities) {
	int i;
	float fitness=0.0;
	for (i=0;i<num_cities-1;i++) {
		fitness+=lookup_distance(tour->city[i]->id,tour->city[i+1]->id);
	}
	// do we count arr[n] --> arr[0] ?
	tour->fitness=fitness;
//	return fitness;
}

/**
 * Generates the nearest neighbor tour based on a random city.
 */
tour_t* create_tour_nn(city_t* city, int num_cities, tour_t* cities) {
	// Set up the cities_visited array; 0 for not visited, 1 for visited.
	char *cities_visited;
	cities_visited = (char *)malloc( num_cities * sizeof(char) );
	memset((void*)cities_visited, 0, sizeof(cities_visited));
	// The tour to be returned.
	tour_t* tour;
	tour = (tour_t*)malloc( sizeof(tour_t) );
	// The next city to place in the tour.
	city_t* next_city;
	// Init to be the city passed into the function
	next_city = city;
	// The first city is city passed.
	tour->city[0] = city;
	cities_visited[ city->id ]=1;

	int i;

	// Iterate through the cities, adding new ones and marking them off.
	for (i=1;i<num_cities;i++) {
		next_city = find_nearest_neighbor(next_city,num_cities,cities,cities_visited);
		tour->city[i]=next_city;
		cities_visited[ next_city->id ]=1;
	}

	// Before returning, set the tour's size.
	tour->size=num_cities;
	return tour;
}

/**
 * Given a city, find its nearest neighbor. The array cities_visited denotes the id of cities
 *  which are available (0) and unavaiable/already visited (1).
 */
city_t* find_nearest_neighbor(city_t* city, int num_cities, tour_t* cities, char* cities_visited) {
	city_t* short_city;
	short_city=(city_t*)malloc( sizeof(city_t) );
	float temp_dist,short_dist;
	temp_dist=short_dist=0.0;
	int i;

	for (i=0;i<num_cities;i++) {
		if (cities->city[i]->id == city->id) {
			continue;
		}
		temp_dist = get_distance_between(cities->city[i]->id,city->id,cities);
		if (  temp_dist < short_dist && cities_visited[i]==0) {
			// If your distance was shorter than the shortest, use this instead.
			short_city = cities->city[i];
			short_dist = temp_dist;
		} else if (short_dist==0 && cities_visited[i]==0) {
			// Otherwise, if not already set, get the first distance as your shortest.
			short_city = cities->city[i];
			short_dist = temp_dist;
		}
	}
	return short_city;
}

/**
 * Given an array of yours and the number of tours in the array, randomly
 * choose one of the tours. The choice is weighted based on the fitness
 * of the function, inversely. In other words, for fitness F1 for tour T1,
 * your probability of receiving tour T1 is (1/F1) / sum( 1/Fi ).
 */
tour_t* roulette_select(tour_t tours[], int num_tours) {
	int i;
	float rand,rand_fit,sum_fitness,temp;
	sum_fitness=0.0;

	// sum up the inverted total fitnesses
	for (i=0;i<num_tours;i++) {
		temp = tours[i].fitness;
		temp = 1.0 / temp;
		sum_fitness+= temp;
	}

	// random float from 0 to 1
	rand=frand();

	// some random point between 0 and top fitness
	rand_fit = sum_fitness * rand;

	for (i=0;i<num_tours;i++) {
		temp = 1.0 / tours[i].fitness;
		if (rand_fit < temp) {
			// If your fitness is in this tour, return it.
			return &tours[i];
		} else {
			// Otherwise, subtract this tour's fitness from sum_fitness and try again.
			rand_fit-=temp;
		}
	}
	// never executes.
	return;
}

int main() {
	// init the cities
	int num_cities = 5;
	tour_t* myCities = loadCities("input/cities1.in");

	// construct the distance table.
	construct_distTable(myCities,num_cities);
}
